#include <chrono>
#include <cstdlib>
#include <exception>
#include <iostream>
#include <sys/stat.h>
#include <vector>

#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>

#include "config.hpp"
#include "errors.hpp"
#include "main_pool.cuh"

using std::cerr;
using std::cout;
using std::endl;
using std::exception;
using std::string;
using std::vector;

int main(int argc, char *argv[])
{
    std::string configfile;
    InConfig config;
    SetDefaultConfig(config);

    if (argc >= 2) {
        for (int iarg = 0; iarg < argc; iarg++) {
            if (std::string(argv[iarg]) == "--config") {      // configuration file
                iarg++;
                configfile = std::string(argv[iarg]);
                try {
                    ReadConfig(configfile, config);
                } catch (const exception &exc) {
                    cout << exc.what() << endl;
                    return 1;
                }
            }
            if (std::string(argv[iarg]) == "-s") {     // the number of streams to use
                iarg++;
                config.nostreams = atoi(argv[iarg]);
            } else if (std::string(argv[iarg]) == "-b") {     // the number of telescopes to accept the data from
                iarg++;
                config.nobeams = atoi(argv[iarg]);
            } else if (std::string(argv[iarg]) == "-t") {     // the number of time sample to average
                iarg++;
                config.timeavg = atoi(argv[iarg]);
            } else if (std::string(argv[iarg]) == "-f") {     // the number of frequency channels to average
                iarg++;
                config.freqavg = atoi(argv[iarg]);
            } else if (std::string(argv[iarg]) == "-n") {    // the number of GPUs to use
                iarg++;
                config.nogpus = atoi(argv[iarg]);
            } else if (std::string(argv[iarg]) == "-o") {    // output directory for the filterbank files
                iarg++;
                struct stat chkdir;
                if (stat(argv[iarg], &chkdir) == -1) {
                    cerr << "Stat error" << endl;
                } else {
                    bool isdir = S_ISDIR(chkdir.st_mode);
                    if (isdir)
                        config.outdir = std::string(argv[iarg]);
                    else
                        cout << "Output directory does not exist! Will use default directory!";
                }
            } else if (std::string(argv[iarg]) == "--gpuid") {
                for (int igpu = 0; igpu < config.nogpus; igpu++) {
                    iarg++;
                    config.gpuids.push_back(atoi(argv[iarg]));
                }
            } else if (std::string(argv[iarg]) == "--ip") {
                for (int iip = 0; iip < config.nogpus; iip++) {
                    iarg++;
                    config.ips.push_back(std::string(argv[iarg]));
                }
            } else if (std::string(argv[iarg]) == "-v") {
                config.verbose = true;
            } else if ((std::string(argv[iarg]) == "-h") || (std::string(argv[iarg]) == "--help")) {
                cout << "Options:\n"
                        << "\t -h --help - print out this message\n"
                        << "\t --config <file name> - configuration file\n"
                        << "\t - THE USE OF FOLLOWING OPTIONS WILL BE PHASED OUT! USE CONFIG FILE INSTEAD!\n"
                        << "\t -b - the number of beams to process\n"
                        << "\t -c - the number of chunks to process\n"
                        << "\t -f - the number of frequency channels to average\n"
                        << "\t -n - the number of GPUs to use\n"
                        << "\t -o <directory> - output directory\n"
                        << "\t -s - the number of CUDA streams per GPU to use\n"
                        << "\t -t - the number of time samples to average\n"
                        << "\t -v - use verbose mode\n"
                        << "\t --gpuid - GPU IDs to use - the number must be the same as 'n'\n"
                        << "\t --ip - IPs to listen to - the number must be the same as 'n'\n\n";
                exit(EXIT_SUCCESS);
            }
        }

    }

    if (config.verbose) {
        cout << "Starting up. This may take few seconds..." << endl;
        PrintConfig(config);
    }

    MainPool workpool(config);

    hipDeviceReset();

    return 0;
}
