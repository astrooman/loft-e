#include "hip/hip_runtime.h"
#include <algorithm>
#include <bitset>
#include <iostream>
#include <fstream>
#include <memory>
#include <mutex>
#include <queue>
#include <sstream>
#include <thread>
#include <utility>
#include <vector>

#include <boost/array.hpp>
#include <boost/asio.hpp>
#include <boost/bind.hpp>
#include <hipfft/hipfft.h>
#include <hip/hip_runtime.h>
#include <numa.h>
#include <pthread.h>
#include <thrust/device_vector.h>

#include "buffer.cuh"
#include "config.hpp"
#include "dedisp/dedisp.hpp"
#include "dedisp/DedispPlan.hpp"
#include "errors.hpp"
#include "filterbank.hpp"
#include "get_mjd.hpp"
#include "heimdall/pipeline.hpp"
#include "kernels.cuh"
#include "obs_time.hpp"
#include "pdif.hpp"
#include "pool_multi.cuh"

#include <inttypes.h>
#include <errno.h>
#include <netdb.h>
#include <arpa/inet.h>
#include <netinet/in.h>
#include <sys/socket.h>
#include <sys/wait.h>
#include <unistd.h>
#include <signal.h>

using std::cerr;
using std::cout;
using std::endl;
using std::mutex;
using std::ostringstream;
using std::pair;
using std::queue;
using std::string;
using std::thread;
using std::unique_ptr;
using std::vector;

#define HEADER 32

mutex cout_guard;

/* ########################################################
TODO: Too many copies - could I use move in certain places?
#########################################################*/

/*####################################################
IMPORTANT: from what I seen in the system files:
There is only one NUMA node.
6 (sic!) physical cores
####################################################*/

int power2factor(unsigned int inbytes) {
    if ((inbytes % 2) != 0)
        return 1;      // don't even  bother with odd numbers

    int factor = 4;

    while ((inbytes % factor) == 0) {
        factor *= 2;
    }

    return factor / 2;
};

Oberpool::Oberpool(InConfig config) : ngpus(config.ngpus)
{

    for (int ii = 0; ii < ngpus; ii++) {
        gpuvector.push_back(unique_ptr<GPUpool>(new GPUpool(ii, config)));
    }

    for (int ii = 0; ii < ngpus; ii++) {
        threadvector.push_back(thread(&GPUpool::Initialise, std::move(gpuvector[ii])));
    }

}

Oberpool::~Oberpool(void)
{
    for (int ii = 0; ii < ngpus; ii++) {
        threadvector[ii].join();
    }

}

bool GPUpool::working_ = true;

GPUpool::GPUpool(int id, InConfig config) : accumulate_(config.accumulate),
                                            avgfreq_(config.freqavg),
                                            avgtime_(config.timesavg),
                                            fftpoints_(config.fftsize),
                                            gpuid_(config.gpuids[id]),
                                            headlen_(config.headlen),
                                            inbits_(config.inbits),
                                            nopols_(config.npol),
                                            nostokes_(config.stokes),
                                            nostreams_(config.streamno),
                                            poolid_(id),
                                            ports_(config.ports[id]),
                                            vdiflen_(config.vdiflen),
                                            verbose_(config.verbose)


{
    availthreads_ = min(nostreams_ + 1, thread::hardware_concurrency());

    config_ = config;

    if (verbose_) {
        cout_guard.lock();
        cout << "Starting GPU pool " << gpuid_ << endl;
        cout << "This may take few seconds..." << endl;
	    cout.flush();
        cout_guard.unlock();
    }
}

GPUpool::~GPUpool(void)
{
    // TODO: join the processing threads

    cudaCheckError(hipFree(dscaled_));
    cudaCheckError(hipFree(dpower_));

    for (int istoke = 0; istoke < nostokes_; istoke++) {
        cudaCheckError(hipFree(hdscaled_[istoke]));
        cudaCheckError(hipFree(hdpower_[istoke]));
    }

    cudaCheckError(hipFree(dfft_));
    cudaCheckError(hipFree(dunpacked_));
    cudaCheckError(hipFree(dinpol_));

    for (int ipol = 0; ipol < nopols_; ipol++) {
        cudaCheckError(hipFree(hdunpacked_));
        cudaCheckError(hipFree(hdinpol_[ipol]));
        cudaCheckError(hipHostFree(inpol_[ipol]));
    }

    cudaCheckError(hipHostFree(inpol_));

    delete [] hdscaled_;
    delete [] hdpower_;
    delete [] hdunpacked_;
    delete [] hdinpol_;
}

void GPUpool::Initialise(void)
{

    noports_ = ports_.size();

    signal(SIGINT, GPUpool::HandleSignal);
    cudaCheckError(hipSetDevice(gpuid_));

    cpu_set_t cpuset;
    CPU_ZERO(&cpuset);
    // * 3 as there are 3 cores available for each telescope/GPU
    CPU_SET((int)(gpuid_) * 3, &cpuset);
    int retaff = pthread_setaffinity_np(pthread_self(), sizeof(cpu_set_t), &cpuset);

    if (retaff != 0) {
        cout_guard.lock();
        cerr << "Error setting thread affinity for the GPU pool " << gpuid_ << endl;
        cout_guard.unlock();
    }

    if(verbose_) {
        cout_guard.lock();
        cout << "GPU pool for device " << gpuid_ << " running on CPU " << sched_getcpu() << endl;
        cout_guard.unlock();
    }

    // STAGE: memory
    if (verbose_)
        cout << "Initialising the memory..." << endl;

    inpolbufsize_ = 2 * accumulate_ * vdiflen_;
    inpolgpusize_ = accumulate_ * vdiflen_ * nostreams_;
    int unpackfactor = 8 / inbits_;
    unpackedsize_ = accumulate_ * vdiflen_ * nostreams_ * unpackfactor;
    powersize_ = accumulate_ * vdiflen_ * nostreams_ * unpackfactor;
    // averaging will happen in either power or scale kernel - will be decided later
    scaledsize_ = accumulate_ * vdiflen_ * nostreams_ * unpackfactor / avgfreq_ / avgtime_;

    hdinpol_ = new unsigned char*[nopols_];
    hdunpacked_ = new float*[nopols_];
    hdpower_ = new float*[nostokes_];
    hdscaled_ = new unsigned char*[nostokes_];
    cudaCheckError(hipHostAlloc((void**)&inpol_, nopols_ * sizeof(unsigned char*), hipHostMallocDefault));

    for (int ipol = 0; ipol < nopols_; ipol++) {
        cudaCheckError(hipHostAlloc((void**)&inpol_[ipol], inpolbufsize_ * sizeof(unsigned char), hipHostMallocDefault));
        cudaCheckError(hipMalloc((void**)&hdinpol_[ipol], inpolgpusize_ * sizeof(unsigned char)));
        cudaCheckError(hipMalloc((void**)&hdunpacked_[ipol], unpackedsize_ * sizeof(float)));      // remember we are unpacking to float
    }

    cudaCheckError(hipMalloc((void**)&dinpol_, nopols_ * sizeof(unsigned char*)));
    cudaCheckError(hipMemcpy(dinpol_, hdinpol_, nopols_ * sizeof(unsigned char*), hipMemcpyHostToDevice));

    cudaCheckError(hipMalloc((void**)&dunpacked_, nopols_ * sizeof(float*)));
    cudaCheckError(hipMemcpy(dunpacked_, hdunpacked_, nopols_ * sizeof(float*), hipMemcpyHostToDevice));

    cudaCheckError(hipMalloc((void**)&dfft_, nopols_ * sizeof(hipfftComplex*)));
    cudaCheckError(hipMemcpy(dfft_, hdfft_, nopols_ * sizeof(hipfftComplex*), hipMemcpyHostToDevice));

    for (int istoke = 0; istoke < nostokes_; istoke++) {
        cudaCheckError(hipMalloc((void**)&hdpower_[istoke], powersize_ * sizeof(float)));
        // TODO: this should really be template-like - we may choose to scale to different number of bits
        cudaCheckError(hipMalloc((void**)&hdscaled_[istoke], scaledsize_ * sizeof(unsigned char)));
    }

    cudaCheckError(hipMalloc((void**)&dpower_, nostokes_ * sizeof(float*)));
    cudaCheckError(hipMemcpy(dpower_, hdpower_, nostokes_ * sizeof(float*), hipMemcpyHostToDevice));

    cudaCheckError(hipMalloc((void**)&dscaled_, nostokes_ * sizeof(unsigned char*)));
    cudaCheckError(hipMemcpy(dscaled_, hdscaled_, nostokes_ * sizeof(unsigned char*), hipMemcpyHostToDevice));

    // TODO: this should really be template-like - we may choose to scale to different number of bits
    filbuffer_ = unique_ptr<Buffer<unsigned char>>(new Buffer<unsigned char>(gpuid_));

    // STAGE: prepare and launch GPU work
    if (verbose_)
        cout << "Launching the GPU..." << endl;

    gpustreams_ = new hipStream_t[nostreams_];
    fftplans_ = new hipfftHandle[nostreams_];
    fftsizes_ = new int[1];
    fftsizes_[0] = fftpoints_;


    for (int igstream = 0; igstream < nostreams_; igstream++) {
        cudaCheckError(hipStreamCreate(&gpustreams_[igstream]));
        cufftCheckError(hipfftPlanMany(&fftplans_[igstream], 1, fftsizes_, NULL, 1, fftpoints_, NULL, 1, fftpoints_, HIPFFT_R2C, fftbatchsize_));
        cufftCheckError(hipfftSetStream(fftplans_[igstream], gpustreams_[igstream]));
    }

    int nokernels = 3;  // unpack, power and scale
    cudablocks_ = new unsigned int[nokernels];
    cudathreads_ = new unsigned int[nokernels];

    // currently limit set to 1024, but can be lowered down, depending on the results of the tests
    sampperthread_ = min(power2factor(accumulate_ * vdiflen_), 1024);
    int needthreads = accumulate_ * vdiflen_ / sampperthread_;
    cudathreads_[0] = min(needthreads, 1024);
    int needblocks = needthreads / cudathreads_[0];
    cudablocks_[0] = min(needblocks, 65536);

    rem_ = needthreads - cudablocks_[0] * cudathreads_[0];

    for (int igstream = 0; igstream < nostreams_; igstream++) {
        gputhreads_.push_back(thread(&GPUpool::DoGpuWork, this, igstream));
    }

    // STAGE: networking
    if (verbose_)
        cout << "Setting up networking..." << endl;

    int netrv;
    addrinfo hints, *servinfo, *tryme;
    memset(&hints, 0, sizeof(hints));
    hints.ai_family = AF_INET;
    hints.ai_socktype = SOCK_DGRAM;
    hints.ai_flags = AI_PASSIVE;

    sockfiledesc_ = new int[noports_];
    recbufs_ = new unsigned char*[noports_];

    for (int iport = 0; iport < noports_; iport++)
        recbufs_[iport] = new unsigned char[vdiflen_ + headlen_];

    ostringstream ssport;
    string strport;

    for (int iport = 0; iport < noports_; iport++) {
        ssport.str("");
        ssport << ports_[iport];
        strport = ssport.str();

        if((netrv = getaddrinfo(strip_[iport].c_str(), strport.c_str(), &hints, &servinfo)) != 0) {
            cout_guard.lock();
            cerr <<  "getaddrinfo() error: " << gai_strerror(netrv) << endl;
            cout_guard.unlock();
        }

        for (tryme = servinfo; tryme != NULL; tryme=tryme->ai_next) {
            if((sockfiledesc_[iport] = socket(tryme->ai_family, tryme->ai_socktype, tryme->ai_protocol)) == -1) {
                cout_guard.lock();
                cerr << "Socket error\n";
                cout_guard.unlock();
                continue;
            }

            if(bind(sockfiledesc_[iport], tryme->ai_addr, tryme->ai_addrlen) == -1) {
                close(sockfiledesc_[iport]);
                cout_guard.lock();
                cerr << "Bind error\n";
                cout_guard.unlock();
                continue;
            }
            break;
        }

        if (tryme == NULL) {
            cout_guard.lock();
            cerr << "Failed to bind to the socket " << ports_[iport] << "\n";
            cout_guard.unlock();
        }
    }

    int bufres{4*1024*1024};    // 4MB

    for (int iport = 0; iport < noports_; iport++) {
        if(setsockopt(sockfiledesc_[iport], SOL_SOCKET, SO_RCVBUF, (char *)&bufres, sizeof(bufres)) != 0) {
            cout_guard.lock();
            cerr << "Setsockopt error on port " << ports_[iport] << endl;
            cerr << "Errno " << errno << endl;
            cout_guard.unlock();
        }
    }

    for (int iport = 0; iport < noports_; iport++)
        receivethreads_.push_back(thread(&GPUpool::ReceiveData, this, iport, ports_[iport]));

}

void GPUpool::DoGpuWork(int stream)
{
    // let us hope one stream will be enough or we will have to squeeze multiple streams into single CPU core
    cpu_set_t cpuset;
    CPU_ZERO(&cpuset);
    CPU_SET((int)gpuid_ * 3 + 1, &cpuset);

    int retaff = pthread_setaffinity_np(gputhreads_[stream].native_handle(), sizeof(cpu_set_t), &cpuset);

    if (retaff != 0) {
        cerr << "Error setting thread affinity for the GPU processing, stream " << stream << endl;
    }

    if (verbose_) {
        cout_guard.lock();
        cout << "Starting worker " << gpuid_ << ":" << stream << " on CPU " << sched_getcpu() << endl;
        cout_guard.unlock();
    }

    cudaCheckError(hipSetDevice(gpuid_));

    while (working_) {

        for (int ipol = 0; ipol < nopols_; ipol++) {
            cudaCheckError(hipMemcpyAsync(hdinpol_[ipol], inpol_[ipol], accumulate_ * vdiflen_, hipMemcpyHostToDevice, gpustreams_[stream]));
        }

        UnpackKernel<<<cudablocks_[0], cudathreads_[0], 0, gpustreams_[stream]>>>(dinpol_, dunpacked_, nopols_, sampperthread_, rem_, accumulate_ * vdiflen_);
        // TODO: test - will cufft handle a 2D array? like dunpacked_?
        // cufftCheckError(hipfftExecR2C(fftplans_[stream], dunpacked_, dfft_, HIPFFT_FORWARD));
        PowerKernel<<<cudablocks_[1], cudathreads_[1], 0, gpustreams_[stream]>>>(dfft_, dpower_);
        ScaleKernel<<<cudablocks_[2], cudathreads_[2], 0, gpustreams_[stream]>>>(dpower_, dscaled_);

    }


}

void GPUpool::HandleSignal(int signum)
{

    cout << "Captured the signal\nWill now terminate!\n";
    working_ = false;
}

void GPUpool::ReceiveData(int portid, int recport)
{
    cpu_set_t cpuset;
    CPU_ZERO(&cpuset);
    // TODO: need to test how much we can squeeze out of the single core
    // might depend on portid if can't pack everything in one core
    CPU_SET((int)(poolid_) * 3, &cpuset);
    // TODO: pass the thread ID properly
    int retaff = pthread_setaffinity_np(receivethreads_[portid].native_handle(), sizeof(cpu_set_t), &cpuset);
    if (retaff != 0) {
        cout_guard.lock();
        cerr << "Error setting thread affinity for receive thread on port " << recport << endl;
        cout_guard.unlock();
    }

    if (verbose_) {
        cout_guard.lock();
        cout << "Receive thread on port " << recport << " running on CPU " << sched_getcpu() << endl;
        cout_guard.unlock();
    }

    sockaddr_storage theiraddr;
    memset(&theiraddr, 0, sizeof(theiraddr));
    socklen_t addrlen;
    memset(&addrlen, 0, sizeof(addrlen));

    const int pack_per_worker_buf = packperbuf_ / nostreams_;
    int numbytes{0};
    short bufidx{0};
    // this will always be an integer
    int frameno{0};
    int refsecond{0};
    // thread ID is used to distinguish between polarisations
    int threadid{0};
    int packcount{0};

    // TODO: be careful which port waits
    if (recport == ports_[0]) {
        unsigned char *tempbuf = recbufs_[0];
        numbytes = recvfrom(sockfiledesc_[0], recbufs_[0], vdiflen_ + headlen_, 0, (struct sockaddr*)&theiraddr, &addrlen);
        starttime_.startepoch = (int)(tempbuf[4] & 0x3f);
        starttime_.startsecond = (int)(tempbuf[3] | (tempbuf[2] << 8) | (tempbuf[1] << 16) | ((tempbuf[0] & 0x3f) << 24));
    }

    while (true) {
        if ((numbytes = recvfrom(sockfiledesc_[portid], recbufs_[portid], vdiflen_ * headlen_, 0, (struct sockaddr*)&theiraddr, &addrlen)) == -1) {
            cout_guard.lock();
            cerr << "Error of recvfrom on port " << recport << endl;
            cerr << "Errno " << errno << endl;
            cout_guard.unlock();
        }
        if (numbytes == 0)
            continue;
        frameno = (int)(recbufs_[portid][7] | (recbufs_[portid][6] << 8) | (recbufs_[portid][5] << 16));
        if (frameno == 0) {
            break;
        } // wait until reaching frame zero of next second before beginnning recording
    }

    while(working_) {
        if ((numbytes = recvfrom(sockfiledesc_[portid], recbufs_[portid], vdiflen_ + headlen_, 0, (struct sockaddr*)&theiraddr, &addrlen)) == -1) {
            cout_guard.lock();
            cerr << "Error of recvfrom on port " << recport << endl;
            cerr << "Errno " << errno << endl;
            cout_guard.unlock();
        }
        if (numbytes == 0)
            continue;
        threadid = (int)(recbufs_[portid][13] | ((recbufs_[portid][12] & 0x02) << 8));
        refsecond = (int)(recbufs_[portid][3] | (recbufs_[portid][2] << 8) | (recbufs_[portid][1] << 16) | ((recbufs_[portid][0] & 0x3f) << 24));
        frameno = (int)(recbufs_[portid][7] | (recbufs_[portid][6] << 8) | (recbufs_[portid][5] << 16));
        bufidx = (((int)frameno / accumulate_) % nostreams_) * accumulate_ + (frameno % accumulate_);
        // frametimes[bufidx] = frameno;
        std::copy(recbufs_[portid] + headlen_, recbufs_[portid] + headlen_ + vdiflen_, inpol_[threadid] + vdiflen_ * bufidx);
        //bufidxarray[bufidx] = true;
        //}
    }
}
